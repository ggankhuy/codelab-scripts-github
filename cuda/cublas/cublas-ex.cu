// First, create a cuBLAS handle:
hipblasStatus_t cublasStat = hipblasCreate(&handle);

// Set the math mode to allow cuBLAS to use Tensor Cores:
cublasStat = hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);

// Allocate and initialize your matrices (only the A matrix is shown):
size_t matrixSizeA = (size_t)rowsA * colsA;
T_ELEM_IN **devPtrA = 0;

hipMalloc((void**)&devPtrA[0], matrixSizeA * sizeof(devPtrA[0][0]));
T_ELEM_IN A  = (T_ELEM_IN *)malloc(matrixSizeA * sizeof(A[0]));

memset( A, 0xFF, matrixSizeA* sizeof(A[0]));
status1 = hipblasSetMatrix(rowsA, colsA, sizeof(A[0]), A, rowsA, devPtrA[i], rowsA);

// ... allocate and initialize B and C matrices (not shown) ...

// Invoke the GEMM, ensuring k, lda, ldb, and ldc are all multiples of 8, 
// and m is a multiple of 4:
cublasStat = hipblasGemmEx(handle, transa, transb, m, n, k, alpha,
                          A, HIP_R_16F, lda,
                          B, HIP_R_16F, ldb,
                          beta, C, HIP_R_16F, ldc, HIP_R_32F, algo);
