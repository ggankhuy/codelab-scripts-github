#include "hip/hip_runtime.h"
/*
Simple vector addition. Mem alloc-d using default hipHostAlloc.
page locked/pinned: never page out to disk. dma copy: pinned buffer -> gpu memory.
pageable (normal): could page out. dma copy: pageable buffer -> pinned buffer -> gpu memory.
*/

#include <stdio.h>

#define N 8192

__global__ void add(int *a, int*b, int *c) {
	int tid = blockIdx.x;
//	if (tid < N) 
	c[tid] = a[tid] + b[tid];
}

int main (void) {
	int *dev_a, *dev_b, *dev_c;
    int *a, *b, *c;

    hipHostAlloc((void**)&a, N * sizeof(*dev_a), hipHostMallocDefault);
    hipHostAlloc((void**)&b, N * sizeof(*dev_a), hipHostMallocDefault);
    hipHostAlloc((void**)&c, N * sizeof(*dev_a), hipHostMallocDefault);

    hipMalloc((void**)&dev_a, N * sizeof(*dev_a));
    hipMalloc((void**)&dev_b, N * sizeof(*dev_b));
    hipMalloc((void**)&dev_c, N * sizeof(*dev_c));

	for (int i = 0; i < N ; i ++ ) {
		a[i]  = i;
		b[i] = i + 200;
        c[i] = 998; 
	}
	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

	add<<<N,1>>> (dev_a, dev_b, dev_c);

    hipDeviceSynchronize();
	hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i+=1000 ) {
        printf("%d: %d + %d = %d\n", i, a[i], b[i], c[i]);
    }

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c);
	return 0;
}
