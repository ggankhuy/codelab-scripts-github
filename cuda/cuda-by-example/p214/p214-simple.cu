#include "hip/hip_runtime.h"
/*
Simple vector addition. Mem alloc-d using default hipMalloc.
*/
#include <stdio.h>

#define N 8192

__global__ void add(int *a, int*b, int *c) {
    //int tid = threadIdx.x * blockIdx.x * blockDim.x;
    int tid = blockIdx.x;
	if (tid < N) 
	    c[tid] = a[tid] + b[tid];
}

int main (void) {
	int *dev_a, *dev_b, *dev_c;
    int *a, *b, *c;

    hipHostAlloc((void**)&a, N * sizeof(int), hipHostMallocWriteCombined|hipHostMallocMapped);
    hipHostAlloc((void**)&b, N * sizeof(int), hipHostMallocWriteCombined|hipHostMallocMapped);
    hipHostAlloc((void**)&c, N * sizeof(int), hipHostMallocWriteCombined|hipHostMallocMapped);

    hipHostGetDevicePointer((void**)&dev_a, a, 0);
    hipHostGetDevicePointer((void**)&dev_b, b, 0);
    hipHostGetDevicePointer((void**)&dev_c, c, 0);

	for (int i = 0; i < N ; i ++ ) {
		a[i]  = i;
		b[i] = i + 200;
        c[i] = 999;
	}

	add<<<N,1>>> (dev_a, dev_b, dev_c);

    hipDeviceSynchronize();
    //hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < N; i+=1000 ) {
		printf("%d: %d + %d = %d\n", i, dev_a[i], dev_b[i], dev_c[i]);
		printf("%d: %d + %d = %d\n", i, a[i], b[i], c[i]);
	}

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c);
	return 0;
}
